#include "stdio.h"
#include "hip/hip_runtime.h"

int main(){
    int iDev = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, iDev);

    printf("Device %d: %s \n", iDev, iProp.name);
    printf("Number of multiprocessors: %d\n", iProp.multiProcessorCount);
    printf("Total amount of shared memory per block: %4.2f KB\n", iProp.sharedMemPerBlock/1024.0);
    printf("Total number of registers available per block: %d\n", iProp.regsPerBlock);
    // printf("Warp size %d\n", warpSize);
    printf("Maximum number of threads per block %d \n", iProp.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor: %d\n", iProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor: %d\n", iProp.maxThreadsPerMultiProcessor/32);
    return EXIT_SUCCESS;
}