#include "hip/hip_runtime.h"
#include "stdio.h"

int main(int argc, char **argv){
    int dev = 0; 
    hipSetDevice(dev);

    unsigned int isize = 1<<16;
    unsigned int nbytes = isize * sizeof(float);

    hipDeviceProp_t deviceProp; 
    hipGetDeviceProperties(&deviceProp, dev);

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB \n", dev, deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));

    float *h_a = (float*)malloc(nbytes);
    
    float *d_a;
    hipMalloc((float**)&d_a, nbytes);

    for(int i = 0; i<isize; i++){
        h_a[i] = 0.5f;
    }

    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    free(h_a);

    hipDeviceReset();
    return EXIT_SUCCESS;

}